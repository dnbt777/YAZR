#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <GL/glew.h>
#include <GL/gl.h>
#include <GL/glu.h>
#include <GL/glut.h>
#include <GLFW/glfw3.h>
#include <iostream>
#include <cmath>
#include <cuda_gl_interop.h>

__managed__ int WIDTH;
__managed__ int HEIGHT;
__managed__ int CHANNELS;
__managed__ GLuint textureID;
__managed__ GLenum target;
__managed__ hipGraphicsResource* cudaResource;
__managed__ unsigned char* d_image;

__device__ float magnitude(float var[3]) {
    return sqrt(var[0]*var[0] + var[1]*var[1] + var[2]*var[2]);
}

__device__ void unitvector(float var[3]) {
    float mag = magnitude(var);

    if (mag != 0.0) {
        var[0] /= mag;
        var[1] /= mag;
        var[2] /= mag;
    } else {
        var[0] = 0.0;
        var[1] = 0.0;
        var[2] = 0.0;
    }
}

extern "C" int cuda_device_check() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        printf("No CUDA-capable device found.\n");
        return -1; // Exit if no device is found
    }

    // Print device properties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0); // Get properties of the first device
    printf("Using device: %s\n", deviceProp.name);

    return 0;
}
// nah opengl now
// __managed__ float d_R[1000 * 1000], d_G[1000*1000], d_B[1000*1000];


// todo - make *tex managed
__global__ void renderKernel(
        unsigned char* screen_tex, int width, int height,
        const float pixel_delta_u0,const float pixel_delta_u1,const float pixel_delta_u2,
        const float pixel_delta_v0,const float pixel_delta_v1,const float pixel_delta_v2,
        const float pixel00_loc0,const float pixel00_loc1,const float pixel00_loc2,
        const float origin0, const float origin1, const float origin2,
        int samples
        ) {
        // int samples_per_pixel, int depth,
        // int hittables_flattened, int num_hittables) { //idk what to do about hittable type
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float pixel_center[3];
    float color[3]; 
    float direction[3];

    float unit_direction[3];
    float magnitude;

    // execute threads from (0,0) to (height*ceil(sqrt(samples_per_pixel)), width*ceil(sqrt(samples_per_pixel)))
    // space has to fit each thread or whatever
    // int sqrt_samples_per_pixel = ceil(sqrt(samples_per_pixel)); // may cause issues affecting memory in outside executed threads...
    // one step at a time. first we just trace a single ray per pixel
    for (int i=0;i<samples;i++) {
        if (row < height && col < width) {
            int idx = (row*width + col) * 3; // RGB - `texture` is [R, G, B, R, G, B, R, G, ...]
            // // // // // // // ray color func here // // // // // // //
            // pixel's center in the viewport
            // uh, I think...
            pixel_center[0] = pixel00_loc0 + (row * pixel_delta_u0) + (col * pixel_delta_v0);
            pixel_center[1] = pixel00_loc1 + (row * pixel_delta_u1) + (col * pixel_delta_v1);
            pixel_center[2] = pixel00_loc2 + (row * pixel_delta_u2) + (col * pixel_delta_v2);

            direction[0] = pixel_center[0] - origin0;
            direction[1] = pixel_center[1] - origin1;
            direction[2] = pixel_center[2] - origin2;

            

            // ray_color func (we have origin and direction, which == ray)

            //for sphere in...
            // hit sphere
            float hit_sphere;
            float sphere_center[3];
            sphere_center[0] = 0.0;
            sphere_center[1] = 0.0;
            sphere_center[2] = -1.0;
            float sphere_radius = 0.5;
            float oc[3];
            oc[0] = sphere_center[0] - origin0;
            oc[1] = sphere_center[1] - origin1;
            oc[2] = sphere_center[2] - origin2;
            float a = direction[0]*direction[0] + direction[1]*direction[1] + direction[2]*direction[2]; //dot
            float b = -2.0 * (
                    direction[0]*oc[0] +
                    direction[1]*oc[1] +
                    direction[2]*oc[2]
                    );
            float c = (
                    oc[0]*oc[0] +
                    oc[1]*oc[1] +
                    oc[2]*oc[2] - sphere_radius*sphere_radius
                    );
            float discriminant = b*b - 4*a*c;
            if (discriminant < 0) {
                hit_sphere = -1.0;
            } else {
                hit_sphere = (-b - sqrt(discriminant)) / (2.0*a);
            }
            
            float t = hit_sphere;

            if (t > 0.0) {
                float N[3];
                N[0] = (origin0 + t*direction[0]) - 0.0;
                N[1] = (origin1 + t*direction[1]) - 0.0;
                N[2] = (origin2 + t*direction[2]) - -1.0;
                unitvector(N); // turns N into its unit vector

                color[0] = 0.5*(N[0] + 1.0);
                color[1] = 0.5*(N[1] + 1.0);
                color[2] = 0.5*(N[2] + 1.0);
            } else {
                // ray_color
                // unit_direction
                magnitude = sqrt(direction[0]*direction[0] + direction[1]*direction[1] + direction[2]*direction[2]);
                unit_direction[0] = direction[0]/magnitude;
                unit_direction[1] = direction[1]/magnitude;
                unit_direction[2] = direction[2]/magnitude;

                float a = 0.5*(unit_direction[1] + 1.0);

                color[0] = (1.0-a) * 1.0 + a * 0.5;
                color[1] = (1.0-a) * 1.0 + a * 0.7;
                color[2] = (1.0-a) * 1.0 + a * 1.0;
            }

            // // // // // // // write_color // // // // // // //
            screen_tex[idx] = color[0];
            screen_tex[idx + 1] = color[1];
            screen_tex[idx + 2] = color[2];

        }
    }
}

__managed__ unsigned char tex;

// instead of doing more samples I can just keep calling this over and over again!
// I just have to initialize R G B to zeroes
// then on each pass, I just add the color
// then at the end average them out..?
// has flaws at high sampling but whatever
extern "C" void render(
        int width, int height,
        const float pixel_delta_u0,const float pixel_delta_u1,const float pixel_delta_u2,
        const float pixel_delta_v0,const float pixel_delta_v1,const float pixel_delta_v2,
        const float pixel00_loc0,const float pixel00_loc1,const float pixel00_loc2,
        const float origin0, const float origin1, const float origin2,
        int samples
        ) {
    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    hipError_t err;
    // size_t size = width * height * sizeof(float);

    // Launch the kernel
    renderKernel<<<numBlocks, threadsPerBlock>>>(
            &tex,
            width, height,
            pixel_delta_u0,
            pixel_delta_u1,
            pixel_delta_u2,
            pixel_delta_v0,
            pixel_delta_v1,
            pixel_delta_v2,
            pixel00_loc0,
            pixel00_loc1,
            pixel00_loc2,
            origin0,
            origin1,
            origin2,
            samples
            );
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
    }

}

 
void displayTexture() {
    glClear(GL_COLOR_BUFFER_BIT);
    printf("cleared\n");
    glBindTexture(GL_TEXTURE_2D, textureID);
    printf("texture bound\n");
    // draw a single quad covering the entire window
    // this is where the texture will be rendered
    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f); glVertex2f(-1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, 1.0f);
    glTexCoord2f(0.0f, 1.0f); glVertex2f(-1.0f, 1.0f);
    glEnd(); // end drawing the quad
    printf("quad drawn\n");
    glutSwapBuffers();
    printf("texture displayed\n");
}



void updateTexture() {
    size_t num_bytes;

    hipGraphicsMapResources(1, &cudaResource, 0);
    printf("resources mapped\n");
    hipGraphicsResourceGetMappedPointer((void**)&d_image, &num_bytes, cudaResource);
    printf("got mapped pointer\n");

    glBindTexture(target, textureID);
    printf("texture bound\n");
    glTexSubImage2D(target, 0, 0, 0, WIDTH, HEIGHT, GL_RGB32F, GL_UNSIGNED_BYTE, d_image);
    printf("subimage2d written\n");
    hipGraphicsUnmapResources(1, &cudaResource, 0);
    printf("resources unmapped");
}



// initialize the *tex, the window, etc given parameters.
// 
extern "C" int initScene(
    int width, int height,
    const float pixel_delta_u0,const float pixel_delta_u1,const float pixel_delta_u2,
    const float pixel_delta_v0,const float pixel_delta_v1,const float pixel_delta_v2,
    const float pixel00_loc0,const float pixel00_loc1,const float pixel00_loc2,
    const float origin0, const float origin1, const float origin2,
    int samples) {

    CHANNELS = 3;
    WIDTH = width;
    HEIGHT = height;
    target = GL_TEXTURE_2D;

    // Create a windowed mode window and its OpenGL context
    int argc = 0; // workaround for not passing argc and argv in via main()
    glutInit(&argc, NULL);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
    glutInitWindowSize(width, height);
    glutCreateWindow("CUDA OpenGL Interop");
    printf("Window created\n");

    // initialize glew
    glewInit();
    printf("glew initialized\n");

    //initOpenGL
    glEnable(target);
    glGenTextures(1, &textureID);
    glBindTexture(target, textureID);
    
    glTexImage2D(target, 0, GL_RGB32F, WIDTH, HEIGHT, 0, GL_RGB32F, GL_UNSIGNED_BYTE, NULL);
    
    glTexParameteri(target, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(target, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    printf("OpenGL initiailized\n");

    //initCUDA
    hipMalloc((void**)&d_image, width * height * CHANNELS * sizeof(unsigned char));
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    renderKernel<<<gridSize, blockSize>>>(
        d_image,
        width, height,
        pixel_delta_u0,
        pixel_delta_u1,
        pixel_delta_u2,
        pixel_delta_v0,
        pixel_delta_v1,
        pixel_delta_v2,
        pixel00_loc0,
        pixel00_loc1,
        pixel00_loc2,
        origin0,
        origin1,
        origin2,
        samples
    );
    hipDeviceSynchronize();
    hipGraphicsGLRegisterImage(&cudaResource, textureID, target, hipGraphicsRegisterFlagsNone);
    printf("cuda initialized\n");

    // other stuff    
    glutDisplayFunc(displayTexture);
    printf("texture display func set\n");
    glutIdleFunc(updateTexture);
    printf("texture update func set\n");
    glutMainLoop();

    hipGraphicsUnregisterResource(cudaResource);
    return 0;
}

